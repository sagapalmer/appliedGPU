
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <stdlib.h>
#include <cmath>

#define DataType double

__global__ void vecAdd(DataType *in1, DataType *in2, DataType *out, int len) {
  //@@ Insert code to implement vector addition here
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < len) {
    out[idx] = in1[idx] + in2[idx];
  }
}

//@@ Insert code to implement timer start
double getTime() {
  struct timeval t1;
  gettimeofday(&t1, NULL);
  return ((double)t1.tv_sec + (double)t1.tv_usec * 1.e-6);
}

//@@ Insert code to implement timer stop
double getElapsedTime(double startTime) {
  return getTime() - startTime;
}


int main(int argc, char **argv) {

  int inputLength;
  int S_seg;

  //CPU
  DataType *hostInput1;
  DataType *hostInput2;
  DataType *hostOutput;
  DataType *resultRef;

  //GPU
  DataType *deviceInput1;
  DataType *deviceInput2;
  DataType *deviceOutput;

  //@@ Insert code below to read in inputLength from args
  inputLength = atoi(argv[1]); //convert char to int
  S_seg = atoi(argv[2]);

  printf("The input length is %d\n", inputLength);

  //@@ Insert code below to allocate Host memory for input and output
  hostInput1 = (DataType *)malloc(inputLength * sizeof(DataType));
  hostInput2 = (DataType *)malloc(inputLength * sizeof(DataType));
  hostOutput = (DataType *)malloc(inputLength * sizeof(DataType));
  resultRef = (DataType *)malloc(inputLength * sizeof(DataType));


  //@@ Insert code below to initialize hostInput1 and hostInput2 to random numbers, and create reference result in CPU
  double startTimeCPU = getTime();
  for (int i = 0; i < inputLength; i++) {
        hostInput1[i] = static_cast<DataType>(rand()) / RAND_MAX;
        hostInput2[i] = static_cast<DataType>(rand()) / RAND_MAX;
        resultRef[i] = hostInput1[i] + hostInput2[i];
  }
  double elapsedTimeCPU = getElapsedTime(startTimeCPU);


  //@@ Insert code below to allocate GPU memory here
  hipMalloc((void **)&deviceInput1, inputLength * sizeof(DataType));
  hipMalloc((void **)&deviceInput2, inputLength * sizeof(DataType));
  hipMalloc((void **)&deviceOutput, inputLength * sizeof(DataType));

  hipStream_t streams[S_seg];
  for (int i = 0; i < S_seg; i++) {
    hipStreamCreate(&streams[i]);
  }




  //@@ Insert code to below to Copy memory to the GPU here
  // double startDataCopyFromHostToDevice = getTime();
  // //cudaMemcpy(deviceInput1, hostInput1, inputLength * sizeof(DataType), cudaMemcpyHostToDevice);
  // //cudaMemcpy(deviceInput2, hostInput2, inputLength * sizeof(DataType), cudaMemcpyHostToDevice);

  // cudaMemcpyAsync(deviceInput1, hostInput1, inputLength * sizeof(DataType), cudaMemcpyHostToDevice);
  // cudaMemcpyAsync(deviceInput2, hostInput2, inputLength * sizeof(DataType), cudaMemcpyHostToDevice);
  // double endDataCopyFromHostToDevice = getElapsedTime(startDataCopyFromHostToDevice);

  // printf("Data copy from host to device: %.6f seconds\n", endDataCopyFromHostToDevice);

  //@@ Initialize the 1D grid and block dimensions here
  int streamSize = inputLength / S_seg;
  int streamBytes = streamSize * sizeof(DataType);

  int TPB = 128;
  int gridSize = (streamSize + TPB - 1) / TPB;

  //@@ Launch the GPU Kernel here


  double startTimeGPU = getTime();
  for (int i = 0; i < S_seg; i++) {
    int offset = i * streamSize;

    hipMemcpyAsync(&deviceInput1[offset], &hostInput1[offset], streamBytes, hipMemcpyHostToDevice, streams[i]);
    hipMemcpyAsync(&deviceInput2[offset], &hostInput2[offset], streamBytes, hipMemcpyHostToDevice, streams[i]);

    vecAdd<<<gridSize, TPB, 0, streams[i]>>>(&deviceInput1[offset], &deviceInput2[offset], &deviceOutput[offset], streamSize);
    hipMemcpyAsync(&hostOutput[offset], &deviceOutput[offset], streamBytes, hipMemcpyDeviceToHost, streams[i]);
  }


  for (int i = 0; i < S_seg; i++) {
    hipStreamSynchronize(streams[i]);
  }

  double elapsedTimeGPU = getElapsedTime(startTimeGPU);

  for (int i = 0; i < S_seg; i++) {
    hipStreamDestroy(streams[i]);
  }

  printf("Total time: %.6f seconds\n", elapsedTimeGPU);

  //@@ Insert code below to compare the output with the reference
  //printf("Elapsed Time for CPU: %.6f seconds\n", elapsedTimeCPU);
  for (int i = 0; i < inputLength; ++i) {
    if (std::abs(hostOutput[i] - resultRef[i]) > 1e-5) {
        fprintf(stderr, "hostOutput and resultRef does not match on index %d: %f (GPU) != %f (CPU)\n", i, hostOutput[i], resultRef[i]);
    }
  }

  //@@ Free the GPU memory here
  hipFree(deviceInput1);
  hipFree(deviceInput2);
  hipFree(deviceOutput);

  //@@ Free the CPU memory here
  free(hostInput1);
  free(hostInput2);
  free(hostOutput);
  free(resultRef);

  return 0;
}
