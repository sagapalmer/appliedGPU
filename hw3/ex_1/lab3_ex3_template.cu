#include "hip/hip_runtime.h"

#include <stdio.h>
#include <sys/time.h>
#include <random>

#define NUM_BINS 4096

__global__ void histogram_kernel(unsigned int *input, unsigned int *bins,
                                 unsigned int num_elements,
                                 unsigned int num_bins) {

//@@ Insert code below to compute histogram of input using shared memory and atomics
extern __shared__ int s_bins[];
int s_idx = threadIdx.x;

int idx = blockIdx.x * blockDim.x + threadIdx.x;

//initialize shared memory
if (threadIdx.x < num_bins)
  s_bins[s_idx] = 0;

//wait for threads to zero out shared memory
__syncthreads();

if (idx < num_elements) {
  int bin = input[idx];
  atomicAdd(&s_bins[bin], 1);
}

//wait for threads to finish
__syncthreads();

if(threadIdx.x < num_bins)
  atomicAdd(&bins[s_idx], s_bins[s_idx]);

}


__global__ void convert_kernel(unsigned int *bins, unsigned int num_bins) {

//@@ Insert code below to clean up bins that saturate at 127
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < num_bins) {
  if (bins[idx] > 127) {
            bins[idx] = 127;
  }
}
}


int main(int argc, char **argv) {
  
  int inputLength;
  unsigned int *hostInput;
  unsigned int *hostBins;
  unsigned int *resultRef;
  unsigned int *deviceInput;
  unsigned int *deviceBins;

  //@@ Insert code below to read in inputLength from args
  inputLength = atoi(argv[1]);

  printf("The input length is %d\n", inputLength);
  
  //@@ Insert code below to allocate Host memory for input and output
  hostInput = (unsigned int *)malloc(inputLength * sizeof(unsigned int));
  hostBins = (unsigned int *)malloc(NUM_BINS * sizeof(unsigned int));
  resultRef = (unsigned int *)calloc(NUM_BINS * sizeof(unsigned int));

  //hipMallocManaged(&input, inputLength * sizeof(unsigned int));
  //hipMallocManaged(&bins, NUM_BINS * sizeof(unsigned int));

  
  //@@ Insert code below to initialize hostInput to random numbers whose values range from 0 to (NUM_BINS - 1)
  std::random_device rd; 
  std::mt19937 gen(rd());
  std::uniform_int_distribution<> randomValue(0, NUM_BINS);
  for (int i = 0; i < inputLength; i++) {
    hostInput[i] = randomValue(gen);
  }

  /*
  for (int i; i < inputLength; i++) {
    hostInput[i] = rand() % NUM_BINS;
  }
  */

 
  //@@ Insert code below to create reference result in CPU
  //memset(resultRef, 0, NUM_BINS * sizeof(unsigned int));
  for (int i = 0; i < inputLength; i++) {
    int bin = hostInput[i];
    if (resultRef[bin] < 127) {
      resultRef[bin] += 1;
    }
  }

  //@@ Insert code below to allocate GPU memory here
  hipMalloc(&deviceInput, inputLength * sizeof(unsigned int));
  hipMalloc(&deviceBins, NUM_BINS * sizeof(unsigned int));

  //@@ Insert code to Copy memory to the GPU here
  hipMemcpy(deviceInput, hostInput, inputLength * sizeof(unsigned int), hipMemcpyHostToDevice);

  //@@ Insert code to initialize GPU results
  hipMemset(deviceBins, 0, NUM_BINS * sizeof(unsigned int));


  //@@ Initialize the grid and block dimensions here
  int TPB = 512;
  int BLOCKS = (inputLength  + TPB - 1) / TPB;


  // set the size of shared memory
  int smemSize = (TPB)*sizeof(unsigned int);

  //@@ Launch the GPU Kernel here
  histogram_kernel<<BLOCKS, TPB, smemSize>>(deviceInput, deviceBins, inputLength, NUM_BINS);
  hipDeviceSynchronize();


  //@@ Initialize the second grid and block dimensions here
  int TPB_2 = 512;
  int BLOCKS_2 = (inputLength  + TPB_2 - 1) / TPB_2;


  //@@ Launch the second GPU Kernel here
  convert_kernel<<BLOCKS_2, TPB_2>>(deviceBins, NUM_BINS);


  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostBins, deviceBins, NUM_BINS * sizeof(unsigned int), hipMemcpyDeviceToHost);


  //@@ Insert code below to compare the output with the reference
  for (int i = 0; i < NUM_BINS; ++i) {
    if (std::abs(hostBins[i] - resultRef[i]) > 1e-5) {
        fprintf(stderr, "hostBins and resultRef does not match on index %d: %f (GPU) != %f (CPU)\n", i, hostBins[i], resultRef[i]);
    }
  }

  //@@ Free the GPU memory here
  hipFree(deviceInput);
  hipFree(deviceBins);

  //@@ Free the CPU memory here
  free(hostInput);
  free(hostBins);
  free(resultRef);

  return 0;
}

